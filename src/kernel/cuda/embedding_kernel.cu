#include "hip/hip_runtime.h"
#include "kernel/cuda/embedding_kernel.cuh"

#define GLOG_USE_GLOG_EXPORT
#include <glog/logging.h>

namespace mllm
{
    namespace kernel
    {
        __global__ void emb_kernel_cuda_fp32(const uint32_t *input_data,
                                             const float *weight_data,
                                             float *output_data,
                                             uint32_t vocab_size,
                                             uint32_t hidden_size,
                                             uint32_t input_size)
        {
            size_t token = blockIdx.x;
            uint32_t token_id = input_data[token];
            for (uint32_t i = threadIdx.x; i < hidden_size; i += blockDim.x)
            {
                output_data[token * hidden_size + i] = weight_data[token_id * hidden_size + i];
            }
        }

        void emb_kernel_cuda(base::Tensor *input,
                             base::Tensor *weight,
                             base::Tensor *output,
                             size_t vocab_size,
                             size_t hidden_size,
                             void *stream)
        {
            CHECK(stream != nullptr) << "CUDA stream is null.";

            auto shape = input->shape();
            size_t seq_len = shape.size() == 2 ? shape[0] : 1;

            uint32_t *input_data = (uint32_t *)input->data();
            float *weight_data = (float *)weight->data();
            float *output_data = (float *)output->data();

            emb_kernel_cuda_fp32<<<seq_len, 128, 0, (hipStream_t)stream>>>(
                input_data,
                weight_data,
                output_data,
                vocab_size,
                hidden_size,
                seq_len);
        }
    }
}